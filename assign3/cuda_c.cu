

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>

__global__ void matrixMult(int *A, int *B, int *C, int N){
    // Calculate the global row and column for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N){
        // def temprary variable for storing sum 
        int tmp = 0;
        for(int i = 0; i < N; i++){
            tmp += A[row * N + i] * B[i * N + col];
        }

        // storing results back
        C[row * N + col] = tmp;
    }
}


// We will check whether our results are reight or not
/* void verify_result(int *A, int *B, int *C, int N){
    int tmp;
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            tmp = 0;
            for(int k = 0; k < N; k++){
                tmp += A[i * N + k] * B[k * N + j];
            }
            // use the assert for checking the final answer
            assert(tmp == C[i * N + j]);
        }
    }
} */

int main(int argc, char **argv){
    
    int N = atoi(argv[1]);
    clock_t    start, end;              /* timing */
    double  elapsed;
    int size = sizeof(double)*N*N ;
 
    
    start = clock();
    // memory allocation in host RAM
    int *h_A, *h_B, *h_C;
    hipHostMalloc((void **) &h_A, size, hipHostMallocDefault);
    hipHostMalloc((void **) &h_B, size, hipHostMallocDefault);
    hipHostMalloc((void **) &h_C, size, hipHostMallocDefault);
    
    for(int i = 0; i < N; ++i)
    {for( int j = 0;  j < N; ++j)
        { h_A[i*N+j] = (double)rand()/RAND_MAX*100;
          h_B[i*N+j] = (double)rand()/RAND_MAX*100;
          h_C[i*N+j] = 0;
        }
    }


    int *dev_A, *dev_B, *dev_C;
    hipMalloc((void **) &dev_A, size);
    hipMalloc((void **) &dev_B, size);
    hipMalloc((void **) &dev_C, size);
    

    // copy A and B from host to device 
    hipMemcpy(dev_A, h_A, sizeof(int)*N*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, h_B, sizeof(int)*N*N, hipMemcpyHostToDevice);



    // Set our CTA and Grid dimensions
    int threads = 16;
    int blocks = (N + threads - 1) / threads;

    // Setup our kernel launch parameters
    dim3 THREADS(threads, threads);
    dim3 BLOCKS(blocks, blocks);

    // Launch our kernel
    matrixMult<<<BLOCKS, THREADS>>>(dev_A, dev_B, dev_C, N);
    hipMemcpy(h_C, dev_C, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

     end   = clock();
      elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;   
      printf(" \n Time taken for size %d is %f \n",N,elapsed);


    // Verify the result
    // verify_result(h_A, h_B, h_C, N);

    // printf("All Results are correct");
    
    // Free allocated memory
    hipFree(h_A);
    hipFree(h_B);
    hipFree(h_C);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    return 0;
}
